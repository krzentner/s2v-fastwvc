#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include "tensor/gpu_dense_tensor.h"
#include "tensor/gpu_sparse_tensor.h"
#include "tensor/cpu_dense_tensor.h"
#include "tensor/t_data.h"
#include "tensor/cu_rand_kernel.h"
#include "tensor/cuda_helper.h"
#include "tensor/gpu_unary_functor.h"
#include "tensor/gpu_binary_functor.h"
#include "tensor/gpu_reduce_kernel.h"
#include "util/mem_holder.h"

namespace gnn
{
	
template<typename Dtype>
TensorTemplate<GPU, DENSE, Dtype>::TensorTemplate() : Tensor(), data(nullptr)
{
}

template<typename Dtype>
TensorTemplate<GPU, DENSE, Dtype>::TensorTemplate(std::vector<size_t> l) : Tensor()
{
	Reshape(l);
}

template<typename Dtype>
TensorTemplate<GPU, DENSE, Dtype>::TensorTemplate(TShape s) : Tensor()
{
	Reshape(s.dims);
}

template<typename Dtype>
void TensorTemplate<GPU, DENSE, Dtype>::Reshape(std::vector<size_t> l)
{
	this->shape.Reshape(l);

	if (this->data == nullptr)
		this->data = std::make_shared< DenseData<GPU, Dtype> >();

	this->data->Resize(this->shape.Count());
}

template<typename Dtype>
MatType TensorTemplate<GPU, DENSE, Dtype>::GetMatType()
{
	return MatType::dense;
}

template<typename Dtype>
MatMode TensorTemplate<GPU, DENSE, Dtype>::GetMatMode()
{
	return MatMode::gpu;
}

template<typename Dtype>
void TensorTemplate<GPU, DENSE, Dtype>::CopyFrom(DTensor<CPU, Dtype>& src)
{
	Reshape(src.shape.dims);
	hipMemcpy(this->data->ptr, src.data->ptr, sizeof(Dtype) * shape.Count(), hipMemcpyHostToDevice);
}

template<typename Dtype>
void TensorTemplate<GPU, DENSE, Dtype>::CopyFrom(DTensor<GPU, Dtype>& src)
{
	Reshape(src.shape.dims);
	hipMemcpy(this->data->ptr, src.data->ptr, sizeof(Dtype) * shape.Count(), hipMemcpyDeviceToDevice);
}

template<typename Dtype>
void TensorTemplate<GPU, DENSE, Dtype>::ShallowCopy(DTensor<GPU, Dtype>& src)
{
	this->shape = src.shape;
	this->data = src.data;
}

template<typename Dtype>
void TensorTemplate<GPU, DENSE, Dtype>::Zeros()
{
	if (shape.Count())
		hipMemset(data->ptr, 0, this->shape.Count() * sizeof(Dtype));
}

template<typename Dtype>
Dtype TensorTemplate<GPU, DENSE, Dtype>::AsScalar()
{
	ASSERT(this->shape.Count() == 1, "can only convert trivial tensor to scalar");
 	Dtype result;
 	hipMemcpy(&result, this->data->ptr, sizeof(Dtype), hipMemcpyDeviceToHost);
 	return result;
}

template<typename Dtype>
void TensorTemplate<GPU, DENSE, Dtype>::SetRandN(Dtype mean, Dtype std)
{
	SetRand(data->ptr, shape.Count(), NormalRandomizer<Dtype>(mean, std));
}

template<typename Dtype>
void TensorTemplate<GPU, DENSE, Dtype>::SetRandU(Dtype lb, Dtype ub)
{
	SetRand(data->ptr, shape.Count(), UniformRandomizer<Dtype>(lb, ub));
}

template<typename Dtype>
void TensorTemplate<GPU, DENSE, Dtype>::Fill(Dtype scalar)
{
	if (scalar == 0)
		this->Zeros();
	else {
		UnaryEngine<GPU>::Exec<UnarySet>(this->data->ptr, this->shape.Count(), scalar);
	}
}

template<typename Dtype>
Dtype TensorTemplate<GPU, DENSE, Dtype>::ASum()
{
	Dtype result;
	WITH_GPUCTX(ctx, {
		result = Cuda_Asum(ctx.cublasHandle, shape.Count(), data->ptr);
	});
	return result;
}

template<typename Dtype>
void TensorTemplate<GPU, DENSE, Dtype>::ArgMax(DTensor<GPU, int>& dst, uint axis)
{
	ASSERT(axis == 0, "not supported for axis > 0 in GPU DENSE Tensor");
	dst.Reshape({this->shape[0]});
	MatColReduce::Exec(dst.data->ptr, this->data->ptr, this->shape[0], this->shape.Count(1), MaxIdxReduce<Dtype>());
}

template<>
void TensorTemplate<GPU, DENSE, double>::MM(DTensor<GPU, double>& a, DTensor<GPU, double>& b, Trans transA, Trans transB, double alpha, double beta)
{
	ASSERT(a.rank() == 2 && b.rank() == 2, "only support mat x mat now");
	size_t m, n, k;
	GetDims(a.rows(), a.cols(), transA, b.rows(), b.cols(), transB, m, n, k);
	
	Reshape({m, n});
	WITH_GPUCTX(ctx, {
		hipblasDgemm(ctx.cublasHandle, 
	                GPU_T(transB), GPU_T(transA), 
	                n, m, k,
	                &alpha, b.data->ptr, b.cols(), a.data->ptr, a.cols(), 
	                &beta, data->ptr, this->cols());
	});	
}

template<>
void TensorTemplate<GPU, DENSE, float>::MM(DTensor<GPU, float>& a, DTensor<GPU, float>& b, Trans transA, Trans transB, float alpha, float beta)
{
	ASSERT(a.rank() == 2 && b.rank() == 2, "only support mat x mat now");
	size_t m, n, k;
	GetDims(a.rows(), a.cols(), transA, b.rows(), b.cols(), transB, m, n, k);
	
	Reshape({m, n});
	WITH_GPUCTX(ctx, {
	    hipblasSgemm(ctx.cublasHandle,
	                GPU_T(transB), GPU_T(transA), 
	                n, m, k,
	                &alpha, b.data->ptr, b.cols(), a.data->ptr, a.cols(), 
	                &beta, data->ptr, this->cols());
	});	
}


template<typename Dtype>
__global__ void CSRMMKernel(Dtype alpha, int* ptr, int* col_idx, Dtype* val, Dtype* dense_data, int src_cols, Dtype* dst, int dst_cols, int numElements)
{
    int offset = blockDim.x * blockIdx.x + threadIdx.x;
    if (offset < numElements)
    {
        int i = offset / dst_cols, j = offset % dst_cols;        
        for (int t = ptr[i]; t < ptr[i + 1]; ++t)
        {
            dst[offset] += alpha * val[t] * dense_data[col_idx[t] * src_cols + j]; 
        }
    }
}

template<typename Dtype>
void TensorTemplate<GPU, DENSE, Dtype>::MM(SpTensor<GPU, Dtype>& a, DTensor<GPU, Dtype>& b, Trans transA, Trans transB, Dtype alpha, Dtype beta)
{
    assert(transB == Trans::N);
	size_t m, n, k;
	GetDims(a.rows(), a.cols(), transA, b.rows(), b.cols(), transB, m, n, k);

	Reshape({m, n});
    this->Scale(beta);
    if (transA == Trans::N)
    {
        int thread_num = min(c_uCudaThreadNum, this->shape.Count());
        int blocksPerGrid = (this->shape.Count() + thread_num - 1) / thread_num;
        CSRMMKernel <<< blocksPerGrid, thread_num, 0, hipStreamPerThread >>> (alpha, a.data->row_ptr, a.data->col_idx, a.data->val, b.data->ptr, b.cols(), this->data->ptr, this->cols(), this->shape.Count());
    } else 
    {
        DTensor<GPU, Dtype> bt(b.shape);
        DTensor<GPU, Dtype> c({m, n});
        WITH_GPUCTX(ctx, {
            Dtype one = 1.0;
            Dtype zero = 0.0;
            Cuda_GeaM(ctx.cublasHandle, hipblasOperation_t::HIPBLAS_OP_T, hipblasOperation_t::HIPBLAS_OP_T, 
                    b.rows(), b.cols(), &one, b.data->ptr, b.cols(), &zero, b.data->ptr, b.cols(), bt.data->ptr, b.rows());
                               
            Cuda_CSRMM(ctx.cusparseHandle, HIPSPARSE_OPERATION_TRANSPOSE, 
                    a.rows(), b.cols(), a.cols(), a.data->nnz, &alpha, 
                    a.data->val, a.data->row_ptr, a.data->col_idx, bt.data->ptr, bt.rows(), &beta, c.data->ptr, c.rows());                
            Cuda_GeaM(ctx.cublasHandle, hipblasOperation_t::HIPBLAS_OP_T, hipblasOperation_t::HIPBLAS_OP_T, 
                    cols(), rows(), &one, c.data->ptr, c.rows(), &zero, c.data->ptr, c.rows(), data->ptr, n);
        });
    }
}

// Copied from https://github.com/torch/cunn/blob/master/SoftMax.cu
template<typename Dtype>
__global__ void cunn_SoftMax_updateOutput_kernel(Dtype *orig_ptr, int batch_size, int dim)
{
    __shared__ Dtype buffer[REDUCE_THREADS + 1];
    Dtype* dst = orig_ptr + blockIdx.x * dim + blockIdx.y;

    int i_start = threadIdx.x;
    int i_end = dim;
    int i_step = blockDim.x;
    Dtype z;
    // max?
    if (i_start < dim)
        buffer[threadIdx.x] = dst[i_start];
    for (int i = i_start; i < i_end; i += i_step)
    {
        z = dst[i];
        if(buffer[threadIdx.x] < z)
            buffer[threadIdx.x] = z;
    }

    __syncthreads();

    // reduce
    if (threadIdx.x == 0)
    {
        z = buffer[0];
        for (int i = 1; i < min(dim, blockDim.x); i++)
        {
            if(z < buffer[i])
                z = buffer[i];
        }
        buffer[REDUCE_THREADS] = z;
    }

    __syncthreads();

    // sum?
    Dtype max_k = buffer[REDUCE_THREADS];
    buffer[threadIdx.x] = 0;
    for (int i = i_start; i < i_end; i += i_step) 
    {
        z = cuda_exp(dst[i] - max_k);
        buffer[threadIdx.x] += z;
        dst[i] = z;
    }

    __syncthreads();

    // reduce
    if (threadIdx.x == 0)
    {
        z = 0;
        for (int i = 0; i < blockDim.x; i++)
            z += buffer[i];
        buffer[REDUCE_THREADS] = z;
    }

    __syncthreads();

    // softmax
    Dtype sum_k = buffer[REDUCE_THREADS];
    for (int i = i_start; i < i_end; i += i_step)
        dst[i] /= sum_k;
}

template<typename Dtype>
void TensorTemplate<GPU, DENSE, Dtype>::Softmax()
{
	ASSERT(this->rank() == 2, "Softmax is assumed to exec on matrix");
    dim3 blocks(this->rows(), 1);
    dim3 threads(REDUCE_THREADS);
    cunn_SoftMax_updateOutput_kernel <<< blocks, threads, 0, hipStreamPerThread >>> (this->data->ptr, this->rows(), this->cols());  
}

template<typename Dtype>
void TensorTemplate<GPU, DENSE, Dtype>::Scale(Dtype scalar)
{
	if (scalar == 0)
	{
		Zeros();
		return;
	}	
	if (scalar != 1)
	{
		UnaryEngine<GPU>::Exec<UnaryScale>(this->data->ptr, this->shape.Count(), scalar);
	}
}

template<typename Dtype>
void TensorTemplate<GPU, DENSE, Dtype>::Mean(DTensor<GPU, Dtype>& a, int axis)
{
	ASSERT(axis == -1, "currently only support global mean");
	Reshape({1});
	MatColReduce::Exec(this->data->ptr, a.data->ptr, 1, a.shape.Count(), SumReduce<Dtype>());
	Scale(1.0 / a.shape.Count());
}

template<typename Dtype>
void TensorTemplate<GPU, DENSE, Dtype>::Add(Dtype scalar)
{
	UnaryEngine<GPU>::Exec<UnaryAdd>(this->data->ptr, this->shape.Count(), scalar);
}

template<typename Dtype>
void TensorTemplate<GPU, DENSE, Dtype>::Axpy(Dtype a, DTensor<GPU, Dtype>& x)
{
	ASSERT(this->shape == x.shape, "shape doesn't match in Axpy");
	WITH_GPUCTX(ctx, {
		Cuda_Axpy(ctx.cublasHandle, this->shape.Count(), &a, x.data->ptr, data->ptr);
	});
}

template<typename Dtype>
__global__ void SpAxpyKernel(Dtype* dst, int* row_ptr, int* col_idx, Dtype* val, int nnz, int n_rows, int n_cols, Dtype alpha)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < nnz)
    {
        int l = 0, r = n_rows - 1, row;
        while (l <= r)
        {
            row = (l + r) / 2;
            if (row_ptr[row] <= i)
            {
                if (row_ptr[row + 1] > i)
                    break;
                else 
                    l = row + 1;
            } else r = row - 1;
        }
        dst[row * n_cols + col_idx[i]] += val[i] * alpha;
    }
}

template<typename Dtype>
void TensorTemplate<GPU, DENSE, Dtype>::Axpy(Dtype a, SpTensor<GPU, Dtype>& x)
{
	ASSERT(this->shape == x.shape, "shape doesn't match in Axpy");
	int thread_num = min(c_uCudaThreadNum, x.data->nnz);
    int blocksPerGrid = (x.data->nnz + thread_num - 1) / thread_num;

    SpAxpyKernel <<< blocksPerGrid, thread_num, 0, hipStreamPerThread >>> (this->data->ptr, x.data->row_ptr, x.data->col_idx, x.data->val, x.data->nnz, this->rows(), this->cols(), a); 
}

template<typename Dtype>
void TensorTemplate<GPU, DENSE, Dtype>::Axpby(Dtype a, DTensor<GPU, Dtype>& x, Dtype b)
{
	ASSERT(this->shape == x.shape, "shape doesn't match in Axpby");
    Scale(b);
    Axpy(a, x); 
}

template<typename Dtype>
__global__ void SparseEleWiseMulKernel(Dtype* dst, int* row_ptr, int* col_idx, Dtype* val, int n_cols, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (i < numElements) 
    {
        int cur_row = i / n_cols; 
        int cur_col = i % n_cols;
        
        int l = row_ptr[cur_row], r = row_ptr[cur_row + 1] - 1, idx;
        while (l <= r)
        {
            idx = (l + r) / 2;
            if (col_idx[idx] < cur_col)
                l = idx + 1;
            else if (col_idx[idx] > cur_col)
                r = idx - 1;
            else {
                dst[i] *= val[idx];
                return; 
            }
        }
        dst[i] = 0;
    }
}

template<typename Dtype>
void TensorTemplate<GPU, DENSE, Dtype>::ElewiseMul(SpTensor<GPU, Dtype>& src)
{
	ASSERT(this->shape == src.shape, "shape doesn't match in ElewiseMul");
	int thread_num = min(c_uCudaThreadNum, this->shape.Count());
    int blocksPerGrid = (this->shape.Count() + thread_num - 1) / thread_num;
    SparseEleWiseMulKernel <<< blocksPerGrid, thread_num, 0, hipStreamPerThread >>> (this->data->ptr, src.data->row_ptr, src.data->col_idx, src.data->val, this->cols(), this->shape.Count());
}

template<typename Dtype>
__global__ void BCastEleWiseMulKernel(Dtype* dst, size_t* dst_shape, Dtype* src, size_t* src_shape, int rank, size_t* offset, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < numElements)
	{
		int src_idx = 0, d_off = i;
		for (int t = rank - 1; t >= 0; --t)
		{
			int cur_coor = d_off % dst_shape[t];
			if (cur_coor < src_shape[t])
				src_idx += cur_coor * offset[t];
			d_off /= dst_shape[t];
		}
		dst[i] *= src[src_idx];
	}
}

template<typename Dtype>
void TensorTemplate<GPU, DENSE, Dtype>::ElewiseMul(DTensor<GPU, Dtype>& src)
{
	if (this->shape == src.shape)
	{
		BinaryEngine<GPU>::Exec<BinaryMul>(this->data->ptr, src.data->ptr, this->shape.Count());
	} else { // require broadcasting
		ASSERT(this->rank() == src.rank(), "broadcasting only support same rank tensors; please do reshape manually");
		for (size_t i = 0; i < this->rank(); ++i)
			if (shape.dims[i] != src.shape.dims[i])
				ASSERT(src.shape.dims[i] == 1, "shape mismatch, broadcasting failed");
		int thread_num = min(c_uCudaThreadNum, this->shape.Count());
    	int blocksPerGrid = (this->shape.Count() + thread_num - 1) / thread_num;

    	std::vector<size_t> offset(rank());
    	for (size_t i = 0; i + 1 < rank(); ++i)
    		offset[i] = src.shape.Count(i + 1);
    	offset[offset.size() - 1] = 1;

    	thrust::device_vector<size_t> src_shape(src.shape.dims.begin(), src.shape.dims.end());
    	thrust::device_vector<size_t> dst_shape(shape.dims.begin(), shape.dims.end());
    	thrust::device_vector<size_t> dev_off(offset.begin(), offset.end());

    	size_t* ss = thrust::raw_pointer_cast(&src_shape[0]);
    	size_t* ds = thrust::raw_pointer_cast(&dst_shape[0]);
    	size_t* p_off = thrust::raw_pointer_cast(&dev_off[0]);

    	BCastEleWiseMulKernel <<< blocksPerGrid, thread_num, 0, hipStreamPerThread >>>(data->ptr, ds, src.data->ptr, ss, rank(), p_off, shape.Count());
	}
}

template<typename Dtype>
void TensorTemplate<GPU, DENSE, Dtype>::Inv()
{
	UnaryEngine<GPU>::Exec<UnaryInv>(this->data->ptr, this->shape.Count());
}

template<typename Dtype>
Dtype TensorTemplate<GPU, DENSE, Dtype>::Norm2()
{
	Dtype result;
	WITH_GPUCTX(ctx, {
		result = Cuda_Norm2(ctx.cublasHandle, this->shape.Count(), data->ptr);
	});
	return result;
}

template<typename Dtype>
void TensorTemplate<GPU, DENSE, Dtype>::Square()
{
	UnaryEngine<GPU>::Exec<UnarySquare>(this->data->ptr, this->shape.Count());
}

template<typename Dtype>
void TensorTemplate<GPU, DENSE, Dtype>::Sqrt()
{
	UnaryEngine<GPU>::Exec<UnarySqrt>(this->data->ptr, this->shape.Count());
}

template class TensorTemplate<GPU, DENSE, float>;
template class TensorTemplate<GPU, DENSE, double>;

///================================ int tensor ===================================

TensorTemplate<GPU, DENSE, int>::TensorTemplate() : data(nullptr)
{

}

void TensorTemplate<GPU, DENSE, int>::Reshape(std::vector<size_t> l)
{
	this->shape.Reshape(l);

	if (this->data == nullptr)
		this->data = std::make_shared< DenseData<GPU, int> >();

    this->data->Resize(this->shape.Count());
}

MatType TensorTemplate<GPU, DENSE, int>::GetMatType()
{
	return MatType::dense;
}

MatMode TensorTemplate<GPU, DENSE, int>::GetMatMode()
{
	return MatMode::gpu;
}

void TensorTemplate<GPU, DENSE, int>::CopyFrom(DTensor<CPU, int>& src)
{
    Reshape(src.shape.dims);
    hipMemcpy(this->data->ptr, src.data->ptr, sizeof(int) * shape.Count(), hipMemcpyHostToDevice);
}

void TensorTemplate<GPU, DENSE, int>::CopyFrom(DTensor<GPU, int>& src)
{
    Reshape(src.shape.dims);
    hipMemcpy(this->data->ptr, src.data->ptr, sizeof(int) * shape.Count(), hipMemcpyDeviceToDevice);
}

void TensorTemplate<GPU, DENSE, int>::ShallowCopy(DTensor<GPU, int>& src)
{
    this->shape = src.shape;
    this->data = src.data;
}

void TensorTemplate<GPU, DENSE, int>::Zeros()
{
    if (shape.Count())
        hipMemset(data->ptr, 0, this->shape.Count() * sizeof(int));
}

void TensorTemplate<GPU, DENSE, int>::Fill(int scalar)
{
    if (scalar == 0)
        this->Zeros();
    else {
        UnaryEngine<GPU>::Exec<UnarySet>(this->data->ptr, this->shape.Count(), scalar);
    }
}

int TensorTemplate<GPU, DENSE, int>::AsScalar()
{
    ASSERT(this->shape.Count() == 1, "can only convert trivial tensor to scalar");
    int result;
    hipMemcpy(&result, this->data->ptr, sizeof(int), hipMemcpyDeviceToHost);
    return result;
}

template class TensorTemplate<GPU, DENSE, int>;

}